#include "image_codec.h"
#include "nvjpeg.h"
#include "utils.cuh"
#include <fstream>
#include <nvtx3/nvToolsExt.h>

hipStream_t stream;
nvjpegHandle_t nv_handle;

nvjpegJpegState_t nvjpeg_decoder_state;

nvjpegEncoderState_t nv_enc_state;
nvjpegEncoderParams_t nv_enc_params;

image_codec::image_codec()
{
    //THREAD SAFE
    //cuda stream that stores order of operations on GPU
    cuda_log(hipStreamCreate(&stream));
    //library handle
    cuda_log(nvjpegCreateSimple(&nv_handle));

    //NOT THREAD SAFE
    //nvjpeg encoding
    cuda_log(nvjpegEncoderStateCreate(nv_handle, &nv_enc_state, stream));
    cuda_log(nvjpegEncoderParamsCreate(nv_handle, &nv_enc_params, stream));

    // set the highest quality
    cuda_log(nvjpegEncoderParamsSetQuality(nv_enc_params, 100, stream));

    //set the type of encoder - progressive for faster data transfer
    cuda_log(nvjpegEncoderParamsSetEncoding(nv_enc_params, nvjpegJpegEncoding_t::NVJPEG_ENCODING_PROGRESSIVE_DCT_HUFFMAN, stream));

    //nvjpeg decoding
    cuda_log(nvjpegJpegStateCreate(nv_handle, &nvjpeg_decoder_state));
}

ImgFormat image_codec::native_format()
{
    return JPEG;
}

ImageInfo image_codec::read_info(std::vector<unsigned char>* img_buffer)
{
    // Info about input file
    // number of channels in image
    int nComponent = 0;
    nvjpegChromaSubsampling_t subsampling;
    //width and height of every channel
    int widths[NVJPEG_MAX_COMPONENT];
    int heights[NVJPEG_MAX_COMPONENT];

    cuda_log(nvjpegGetImageInfo(nv_handle, img_buffer->data(), img_buffer->size(), &nComponent, &subsampling, widths, heights));

    ImageInfo info;
    info.width = widths[0];
    info.height = heights[0];
    if (nComponent == 1)
    {
        info.colorScheme = ImageColorScheme::IMAGE_GRAY;
    }
    else
    {
        info.colorScheme = ImageColorScheme::IMAGE_RGB;
    }

    //for now, need to figure out how to retrieve it
    info.bit_depth = 8;

    return info;
}

void image_codec::encode(std::vector<unsigned char>* img_buffer, matrix* img_matrix, ImageColorScheme colorScheme, unsigned bit_depth)
{
    nvtxRangeId_t nvtx_render_encode_mark = nvtxRangeStartA("render_encode");

    // code taken from example: https://docs.nvidia.com/cuda/nvjpeg/index.html#nvjpeg-encode-examples

    nvjpegImage_t nv_image;
    //Pitch represents bytes per row
    size_t pitch_0_size = img_matrix->width;

    if (colorScheme == ImageColorScheme::IMAGE_RGB)
    {
        // This has to be done, default params are not sufficient
        // source: https://stackoverflow.com/questions/65929613/nvjpeg-encode-packed-bgr
        cuda_log(nvjpegEncoderParamsSetSamplingFactors(nv_enc_params, NVJPEG_CSS_444, stream));

        pitch_0_size *= 3;
    }
    else
    {
        cuda_log(nvjpegEncoderParamsSetSamplingFactors(nv_enc_params, NVJPEG_CSS_GRAY, stream));
    }

    // Fill nv_image with image data, by copying data from matrix to GPU
    // docs about nv_image: https://docs.nvidia.com/cuda/nvjpeg/index.html#nvjpeg-encode-examples
    cuda_log(hipMalloc((void **)&(nv_image.channel[0]), pitch_0_size * img_matrix->height));
    cuda_log(hipMemcpy(nv_image.channel[0], img_matrix->get_arr_interlaced(), pitch_0_size * img_matrix->height, hipMemcpyHostToDevice));
    
    nv_image.pitch[0] = pitch_0_size;

    // Compress image
    if (colorScheme == ImageColorScheme::IMAGE_RGB)
    {
        cuda_log(nvjpegEncodeImage(nv_handle, nv_enc_state, nv_enc_params,
            &nv_image, nvjpegInputFormat_t::NVJPEG_INPUT_RGBI, img_matrix->width, img_matrix->height, stream));   
    }
    else
    {
        cuda_log(nvjpegEncodeYUV(nv_handle, nv_enc_state, nv_enc_params,
            &nv_image, nvjpegChromaSubsampling_t::NVJPEG_CSS_GRAY, img_matrix->width, img_matrix->height, stream));
    }

    // get compressed stream size
    size_t length = 0;
    cuda_log(nvjpegEncodeRetrieveBitstream(nv_handle, nv_enc_state, NULL, &length, stream));
    // get stream itself
    cuda_log(hipStreamSynchronize(stream));
    img_buffer->clear();
    img_buffer->resize(length);
    cuda_log(nvjpegEncodeRetrieveBitstream(nv_handle, nv_enc_state, img_buffer->data(), &length, 0));

    cuda_log(hipStreamSynchronize(stream));

    //clean up
    cuda_log(hipFree(nv_image.channel[0]));

    nvtxRangeEnd(nvtx_render_encode_mark);
}

void image_codec::decode(std::vector<unsigned char>* img_source, matrix* img_matrix, ImageColorScheme colorScheme, unsigned bit_depth)
{
    if (img_matrix->height == 0 || img_matrix->width == 0)
    {
        return;
    }

    // image resize
    size_t pitch = img_matrix->components_num * img_matrix->width;

    // Image buffer 
    unsigned char * deviceImgBuff = NULL;
    cuda_log(hipMalloc(&deviceImgBuff, pitch * img_matrix->height));

    // device image buffer.
    nvjpegImage_t imgDesc;
    imgDesc.channel[0] = deviceImgBuff;
    imgDesc.pitch[0] = (unsigned int)(img_matrix->width * img_matrix->components_num);

    // decode by stages
    cuda_log(nvjpegDecode(nv_handle, nvjpeg_decoder_state, img_source->data(), img_source->size(), NVJPEG_OUTPUT_RGBI, &imgDesc, NULL));

    img_matrix->resize(img_matrix->width, img_matrix->height);
    cuda_log(hipMemcpy(img_matrix->get_arr_interlaced(), deviceImgBuff, pitch * img_matrix->height, hipMemcpyKind::hipMemcpyDeviceToHost));

    //clean up
    cuda_log(hipFree(deviceImgBuff));
}

void image_codec::load_image_file(std::vector<unsigned char>* img_buff, std::string image_filepath)
{
    std::ifstream oInputStream(image_filepath, std::ios::in | std::ios::binary | std::ios::ate);
    if(!(oInputStream.is_open()))
    {
        return;
    }

    // Get the size.
    std::streamsize nSize = oInputStream.tellg();
    oInputStream.seekg(0, std::ios::beg);
    
    img_buff->resize(nSize);
    oInputStream.read((char*)img_buff->data(), nSize);

    oInputStream.close();
}
        
void image_codec::save_image_file(std::vector<unsigned char>* img_buff, std::string image_filepath)
{
    std::ofstream output_file(image_filepath+".jpeg", std::ios::out | std::ios::binary);
    output_file.write((char *)img_buff->data(), img_buff->size());
    output_file.close();
}

image_codec::~image_codec()
{
    if (nv_enc_params != nullptr)
    {
        cuda_log(nvjpegEncoderParamsDestroy(nv_enc_params));
        nv_enc_params = nullptr;
    }

    if (nv_enc_state != nullptr)
    {
        cuda_log(nvjpegEncoderStateDestroy(nv_enc_state));
        nv_enc_state = nullptr;
    }

    if (nv_handle != nullptr)
    {
        cuda_log(nvjpegDestroy(nv_handle));
        nv_handle = nullptr;
    }

    if (stream != nullptr)
    {
        cuda_log(hipStreamDestroy(stream));
        stream = nullptr;
    }
}
