#include "hip/hip_runtime.h"
#include <cuda/std/limits>
#include <nvtx3/nvToolsExt.h>

#include "vertex_tools.h"
#include "image_draw_objects.h"
#include "utils.cuh"

__global__ void kernel_drawVertices(matrix* m, vertex* vertices, unsigned int n_vertices, unsigned char* components, int scale, int offset)
{
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int comp = threadIdx.y;

    if (i >= n_vertices)
        return;

   int x = static_cast<int>(scale * vertices[i].x + offset);
   int y = static_cast<int>(m->height - (scale * vertices[i].y + offset));

    m->get(x, y)[comp] = components[comp];
};

template<typename E>
void draw_vertices(matrix_color<E>* m, std::vector<vertex>* vertices, E vertex_color, int scale, int offset)
{
    size_t max_block_length = (1024 / m->components_num);

    int block_length = min(max_block_length, vertices->size());

    int block_num = (int)((vertices->size() / block_length)+1);

    dim3 blockSize(block_length, m->components_num);

    vertex* d_vector;
    const unsigned d_vector_bytes = vertices->size() * sizeof(vertex);

    matrix* d_m;
    const unsigned d_m_bytes = sizeof(matrix);

    unsigned char* d_arr_interlaced;
    const unsigned d_arr_interlaced_bytes = m->size_interlaced();
    
    unsigned char* d_vals;
    const unsigned d_vals_bytes = m->components_num * sizeof(unsigned char);

    char* d_membuf;
    cuda_log(hipMalloc(
        &d_membuf,
        d_vector_bytes +
        d_m_bytes +
        d_arr_interlaced_bytes +
        d_vals_bytes
    ));

    unsigned mem_offset = 0;

    d_vector = (vertex*)(d_membuf + mem_offset);
    mem_offset += d_vector_bytes;

    d_m = (matrix*)(d_membuf + mem_offset);
    mem_offset += d_m_bytes;

    d_arr_interlaced = (unsigned char*)(d_membuf + mem_offset);
    mem_offset += d_arr_interlaced_bytes;

    d_vals = (unsigned char*)(d_membuf + mem_offset);
    mem_offset += d_vals_bytes;

    transferMatrixToDevice(d_m, d_arr_interlaced, m);

    unsigned char* h_vals = new unsigned char[m->components_num];

    m->element_to_c_arr(h_vals, vertex_color);
    cuda_log(hipMemcpy(d_vals, h_vals, d_vals_bytes, hipMemcpyHostToDevice));
    cuda_log(hipMemcpy(d_vector, vertices->data(), d_vector_bytes, hipMemcpyHostToDevice));

    kernel_drawVertices<<<block_num, blockSize>>>(d_m, d_vector, vertices->size(), d_vals, scale, offset);

    cuda_log(hipDeviceSynchronize());

    transferMatrixDataToHost(m, d_m, false);

    cuda_log(hipFree(d_membuf));
    
    delete h_vals;
};

__device__ double atomicMin_double(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(fmin(val, __longlong_as_double(assumed))));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void kernel_drawPolygon(matrix* m, matrix_coord screen_min, matrix_coord screen_max, unsigned char* polygon_color, vertex screen_v1, vertex screen_v2, vertex screen_v3, double* zbuffer = nullptr)
{
    matrix_coord i_curr(
        threadIdx.x + blockDim.x * blockIdx.x + screen_min.x,
        threadIdx.y + blockDim.y * blockIdx.y + screen_min.y
    );

    if (i_curr.x >= screen_max.x || i_curr.y >= screen_max.y)
        return;

    vertex baryc = get_barycentric_coords(i_curr, screen_v1, screen_v2, screen_v3);

    if ((baryc.x >= 0) && (baryc.y >= 0) && (baryc.z >= 0))
    {
        //z-buffer check, if available
        if (zbuffer != nullptr)
        {
            int interlaced_index = i_curr.y * m->width + i_curr.x;
            double curr_z = (baryc.x * screen_v1.z + baryc.y * screen_v2.z + baryc.z * screen_v3.z);
            atomicMin_double(zbuffer + interlaced_index, curr_z);

            if (zbuffer[interlaced_index] == curr_z)
            {
                for (unsigned i = 0; i < m->components_num; i++)
                {
                    m->get(i_curr.x, i_curr.y)[i] = polygon_color[i];
                }
            }
        }
        else
        {
            for (unsigned i = 0; i < m->components_num; i++)
            {
                m->get(i_curr.x, i_curr.y)[i] = polygon_color[i];
            }
        }
    }
};

__global__ void kernel_fill(double* arr, unsigned size, double val)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;;

    if (i >= size)
        return;

    arr[i] = val;
}

template<typename E>
void draw_polygon(matrix_color<E>* img, E polyg_color, vertex v1, vertex v2, vertex v3)
{
    matrix_coord min(0,0);
    matrix_coord max(0,0);

    calc_triangle_boundaries(min, max, v1, v2, v3, *img);

    if (max.x <= min.x || max.y <= min.y)
    {
        return;
    }

    //CUDA-SPECIFIC
    unsigned poly_width = max.x - min.x;
    unsigned poly_height = max.y - min.y;

    unsigned blocksize_1d = 32; //32*32 = 1024 = max blocksize

    unsigned blocknum_x = (unsigned)((poly_width/blocksize_1d) +1);
    unsigned blocknum_y = (unsigned)((poly_height/blocksize_1d) +1);

    dim3 blocksize(blocksize_1d, blocksize_1d);
    dim3 blocknum(blocknum_x, blocknum_y);

    matrix* d_m;
    const unsigned d_m_bytes = sizeof(matrix);

    unsigned char* d_arr_interlaced;
    const unsigned d_arr_interlaced_bytes = img->size_interlaced();

    unsigned char* d_vals;
    const unsigned d_vals_bytes = img->components_num * sizeof(unsigned char);

    char* d_membuf;
    cuda_log(hipMalloc(
        &d_membuf,
        d_m_bytes +
        d_arr_interlaced_bytes +
        d_vals_bytes
    ));

    unsigned mem_offset = 0;

    d_m = (matrix*)d_membuf;
    mem_offset += d_m_bytes;

    d_arr_interlaced = (unsigned char*)(d_membuf + mem_offset);
    mem_offset += d_arr_interlaced_bytes;

    d_vals = (unsigned char*)(d_membuf += mem_offset);
    mem_offset += d_vals_bytes;
    
    transferMatrixToDevice(d_m, d_arr_interlaced, img);

    unsigned char* h_vals = new unsigned char[img->components_num];

    img->element_to_c_arr(h_vals, polyg_color);
    cuda_log(hipMemcpy(d_vals, h_vals, d_vals_bytes, hipMemcpyHostToDevice));

    kernel_drawPolygon<<<blocknum, blocksize>>>(d_m, matrix_coord(min.x, min.y), matrix_coord(max.x, max.y), d_vals, v1, v2, v3);

    cuda_log(hipDeviceSynchronize());

    transferMatrixDataToHost(img, d_m, false);
    cuda_log(hipFree(d_membuf));
    delete h_vals;
};

/// @brief Draw polygons and paint them using random values.
/// @param m image matrix
/// @param vertices raw vertices
/// @param polygons raw polygons
/// @param n_vert number of provided vertices 
/// @param n_poly number of provided polygons
/// @param scale how much to scale polygons
/// @param offset how much to offset polygons
/// @param curState used for random numbers generator
/// @param seed used for random numbers generator
__global__ void kernel_drawPolygonsFilled(matrix* m, vertex* vertices, polygon* polygons, unsigned n_vert, unsigned n_poly, double scaleX, double scaleY, double* zbuffer, unsigned char* c_polyg_color_buffer, unsigned char* modelColor)
{
    //polygon index
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i >= n_poly)
    {
        return;
    }

    polygon* curr_poly = polygons + i;

    vertex* poly_v1 = vertices + (curr_poly->vertex_index1-1);
    vertex* poly_v2 = vertices + (curr_poly->vertex_index2-1);
    vertex* poly_v3 = vertices + (curr_poly->vertex_index3-1);

    vertex poly_vec1;
    vertex poly_vec2;

    poly_vertices_to_vectors(*poly_v1, *poly_v2, *poly_v3, poly_vec1, poly_vec2);

    vertex n = normal(poly_vec1, poly_vec2);

    vertex camera_vec(0.0, 0.0, 1.0);
    double d = dot(n, camera_vec);
    double viewing_angle_cosine = d/(length(n)*length(camera_vec));

    unsigned char* c_polyg_color = c_polyg_color_buffer + i * m->components_num;

    if (viewing_angle_cosine >= 0)
    {
        return;
    }

    for (size_t i = 0; i < m->components_num; i++)
    {
        c_polyg_color[i] = (unsigned char)(-1 * modelColor[i] * viewing_angle_cosine + 0.5);
    }

    matrix_coord img_center((unsigned)(m->width/2), (unsigned)(m->height/2));
 
    //retrieve polygon's vertices and scale them
    vertex screen_v1{
        scaleX * poly_v1->x / poly_v1->z + img_center.x,
        m->height - (scaleY * poly_v1->y / poly_v1->z + img_center.y),
        poly_v1->z
    };
    vertex screen_v2{
        scaleX * poly_v2->x / poly_v2->z + img_center.x,
        m->height - (scaleY * poly_v2->y / poly_v2->z + img_center.y),
        poly_v2->z
    };
    vertex screen_v3{
        scaleX * poly_v3->x / poly_v3->z + img_center.x,
        m->height - (scaleY * poly_v3->y/ poly_v3->z + img_center.y),
        poly_v3->z
    };

    //printf("%d,%f,%f,%f,%f,%f,%f,%f,%f,%f,%f,%u\n", i, poly_v1.x, poly_v1.y, poly_v1.z, poly_v2.x, poly_v2.y, poly_v2.z, poly_v3.x, poly_v3.y, poly_v3.z, viewing_angle_cosine, c_polyg_color[0]);

    //calculate rectangular boundary of the triangle
    matrix_coord screen_min(0,0);
    matrix_coord screen_max(0,0);

    calc_triangle_boundaries(screen_min, screen_max, screen_v1, screen_v2, screen_v3, *m);

    if (screen_max.x <= screen_min.x || screen_max.y <= screen_min.y)
    {
        return;
    }

    //DRAW TRIANGLE

    //cuda indices
    unsigned poly_width = screen_max.x - screen_min.x;
    unsigned poly_height = screen_max.y - screen_min.y;

    unsigned total_submatrix_size = poly_width * poly_height;

    unsigned poly_total_blocksize = 32;
    if (total_submatrix_size >= 4480)
    {
        poly_total_blocksize = 128;
    }

    if (total_submatrix_size >= 8960)
    {
        poly_total_blocksize = 256;
    }

    if (total_submatrix_size >= 17920)
    {
        poly_total_blocksize = 512;
    }

    if (total_submatrix_size >= 35840)
    {
        poly_total_blocksize = 1024;
    }

    unsigned blocksize_1d = (unsigned)sqrtf(poly_total_blocksize);

    unsigned blocknum_x = (unsigned)((poly_width/blocksize_1d) +1);
    unsigned blocknum_y = (unsigned)((poly_height/blocksize_1d) +1);

    dim3 blocksize(blocksize_1d, blocksize_1d);
    dim3 blocknum(blocknum_x, blocknum_y);

    hipStream_t s;
    cuda_log(hipStreamCreateWithFlags(&s, hipStreamNonBlocking));

    kernel_drawPolygon<<<blocknum, blocksize, 0, s>>>(m, screen_min, screen_max, c_polyg_color, screen_v1, screen_v2, screen_v3, zbuffer);
    cuda_log(hipGetLastError());
};

template <typename E>
inline void draw_polygons_filled(matrix_color<E> *img, std::vector<vertex> *vertices, std::vector<polygon> *polygons, double scaleX, double scaleY, unsigned char* modelColor)
{
    nvtxRangeId_t nvtx_render_mark = nvtxRangeStartA("render_draw");

    //CUDA-SPECIFIC
    //Итерируемся по полигонам. На каждой такой итерации - получаем нужные вершины и конвертируем в координаты экрана.
    //Затем создаем квадрат в пределах этих вершин и итерируемся по каждому пикселю в квадрате, закрашивая либо не закрашивая по пути
    //основные индексы: i полигонов, x и y итерации по квадрату
    
    unsigned blocksize = 32;
    if (polygons->size() >= 4480)
    {
        blocksize = 128;
    }

    if (polygons->size() >= 8960)
    {
        blocksize = 256;
    }

    if (polygons->size() >= 17920)
    {
        blocksize = 512;
    }
/*
    if (polygons->size() >= 35840)
    {
        blocksize = 1024;
    }
*/

    nvtxRangeId_t nvtx_render_memory_to_mark = nvtxRangeStartA("render_draw_memory_to_gpu");
    double* d_zbuffer;
    const unsigned d_zbuffer_bytes = img->size() * sizeof(double);

    vertex* d_vertices;
    const unsigned d_vertices_bytes = vertices->size() * sizeof(vertex);

    polygon* d_polygons;
    const unsigned d_polygons_bytes = polygons->size() * sizeof(polygon);

    matrix* d_m;
    const unsigned d_m_bytes = sizeof(matrix);

    unsigned char* d_arr_interlaced;
    const unsigned d_arr_interlaced_bytes = img->size_interlaced();

    unsigned char* c_polyg_color_buffer;
    const unsigned c_polyg_color_buffer_bytes = polygons->size() * img->components_num;

    unsigned char* d_modelColor;
    const unsigned d_modelColor_bytes = img->components_num * sizeof(unsigned char);

    char* d_membuf;
    cuda_log(hipMalloc(
        &d_membuf,
        d_zbuffer_bytes +
        d_vertices_bytes +
        d_polygons_bytes +
        d_m_bytes + 
        d_arr_interlaced_bytes + 
        c_polyg_color_buffer_bytes +
        d_modelColor_bytes
    ));

    unsigned mem_offset = 0;

    d_zbuffer = (double*)(d_membuf + mem_offset);
    mem_offset += d_zbuffer_bytes;

    d_vertices = (vertex*)(d_membuf + mem_offset);
    mem_offset += d_vertices_bytes;

    d_polygons = (polygon*)(d_membuf + mem_offset);
    mem_offset += d_polygons_bytes;

    d_m = (matrix*)(d_membuf + mem_offset);
    mem_offset += d_m_bytes;

    d_arr_interlaced = (unsigned char*)(d_membuf + mem_offset);
    mem_offset += d_arr_interlaced_bytes;

    c_polyg_color_buffer = (unsigned char*)(d_membuf + mem_offset);
    mem_offset += c_polyg_color_buffer_bytes;

    d_modelColor = (unsigned char*)(d_membuf + mem_offset);
    mem_offset += d_modelColor_bytes;

    transferMatrixToDevice(d_m, d_arr_interlaced, img);

    cuda_log(hipMemcpy(d_vertices, vertices->data(), d_vertices_bytes, hipMemcpyHostToDevice));
    cuda_log(hipMemcpy(d_polygons, polygons->data(), d_polygons_bytes, hipMemcpyHostToDevice));
    cuda_log(hipMemcpy(d_modelColor, modelColor, d_modelColor_bytes, hipMemcpyHostToDevice));

    nvtxRangeEnd(nvtx_render_memory_to_mark);

    unsigned zbuf_total_blocksize = 32;
    if (img->size() >= 4480)
    {
        zbuf_total_blocksize = 128;
    }

    if (img->size() >= 8960)
    {
        zbuf_total_blocksize = 256;
    }

    if (img->size() >= 17920)
    {
        zbuf_total_blocksize = 512;
    }

    if (img->size() >= 35840)
    {
        zbuf_total_blocksize = 1024;
    }

    kernel_fill<<<(unsigned)(img->size()/zbuf_total_blocksize +1), zbuf_total_blocksize>>>(d_zbuffer, img->size(), std::numeric_limits<double>().max());
    cuda_log(hipGetLastError());

    unsigned blocknum = (unsigned)((polygons->size() / blocksize) + 1);

    //by default the limit on the number of simultaneous kernel launches is imposed 
    cuda_log(hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, polygons->size()));

    kernel_drawPolygonsFilled<<<blocknum, blocksize>>>(d_m, d_vertices, d_polygons, vertices->size(), polygons->size(), scaleX, scaleY, d_zbuffer, c_polyg_color_buffer, d_modelColor);

    cuda_log(hipGetLastError());
    cuda_log(hipDeviceSynchronize());

    nvtxRangeId_t nvtx_render_memory_from_mark = nvtxRangeStartA("render_draw_memory_from_gpu");
    transferMatrixDataToHost(img, d_m, false);

    cuda_log(hipFree(d_membuf));
    nvtxRangeEnd(nvtx_render_memory_from_mark);

    nvtxRangeEnd(nvtx_render_mark);
}

#include "_image_draw_objects_instances.h"
